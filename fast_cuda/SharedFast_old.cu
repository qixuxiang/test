
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime_api.h>
#include <timer.h>
#include <string>
#include "corner.h"
#include "fast_cuda.h"

int main( void )
{
	using namespace std;
	using namespace cv;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEvent_t kernel_start;
	hipEventCreate(&kernel_start);

    const int threshold=20;
    string filename="zazhi.jpg";
	Mat image;
	image = cv::imread(filename,0);   // Read the file
	if(! image.data )                              // Check for invalid input
	{
		cout <<  "Could not open or find the image" << std::endl ;
		return -1;
	}
	hipEventRecord(start);

	uchar* d_data;                // create a pointer
	size_t imSize=image.cols*image.rows;
	Corner* h_corner=new Corner[imSize];
	Corner* d_corner;
	checkCudaErrors(hipMalloc((void**) &d_corner,sizeof(Corner)*imSize));
	checkCudaErrors(hipMalloc((void**) &d_data, sizeof(uchar)*imSize)); // create memory on the gpu and pass a pointer to the host
	checkCudaErrors(hipMemcpy(d_data, image.data, sizeof(uchar)*imSize, hipMemcpyHostToDevice));// copy from the image data to the gpu memory you reserved
	dim3 blocksize(16,16);
	dim3 gridsize((image.cols-1)/blocksize.x+1, (image.rows-1)/blocksize.y+1, 1);
	//hipEventRecord(start);
	hipEventRecord(kernel_start);
	fast<<<gridsize,blocksize>>>(d_data, image.cols, image.rows,d_corner,gridsize.x,gridsize.y,threshold); // processed data on the gpu
	//checkCudaErrors(hipDeviceSynchronize());
	//hipEventRecord(stop);	
	//hipEventSynchronize(stop);
	nms<<<gridsize,blocksize>>>(d_data,d_corner,image.cols,image.rows);
	checkCudaErrors(hipMemcpy(h_corner,d_corner,sizeof(Corner)*imSize,hipMemcpyDeviceToHost));

	hipEventRecord(stop);	
	hipEventSynchronize(stop);

	float elptime;
	hipEventElapsedTime(&elptime,start,stop);

	float elptime2;
	hipEventElapsedTime(&elptime2,kernel_start,stop);

	//show the corner in the image
	Mat image_color = imread(filename,1);
    int point=0;
	for(int i=0;i<imSize;i++)
	{
		if(h_corner[i].set!=0)
		{
			int x=i%image.cols;
			int y=i/image.cols;
			circle(image_color,Point(x,y),1,Scalar(0,255,0),-1,8,0);
			point++;
		}
	}
	cout<<"points:"<<point<<endl;
	cout<<"Elapsed time:"<<elptime<<"ms"<<endl;
	cout<<"Elapsed kernel time:"<<elptime2<<"ms"<<endl;

	//printf("%x\n",0x7|((10>1)<<3));
	//cout<<"the size of: "<<sizeof(corner)<<endl;
	namedWindow( "Display window", WINDOW_AUTOSIZE );// Create a window for display.
	imshow( "Display window", image_color );                   // Show our image inside it.
	waitKey(0);                                          // Wait for a keystroke in the window
	delete[] h_corner;
	hipFree(d_corner);
	hipFree(d_data);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}
